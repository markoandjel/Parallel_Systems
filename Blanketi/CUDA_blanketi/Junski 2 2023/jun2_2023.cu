#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include ""
using namespace std;


__device__ void racunaj(float* dev_niz, float* dev_b)
{
	dev_b[blockDim.x * blockIdx.x + threadIdx.x] = (dev_niz[threadIdx.x] * dev_niz[threadIdx.x+1] * dev_niz[threadIdx.x+2]) /
		(dev_niz[threadIdx.x] + dev_niz[threadIdx.x + 1] + dev_niz[threadIdx.x + 2]);
}
__global__ void kernel(float* dev_a, float*dev_b)
{
	__shared__ float dev_niz[256];
	dev_niz[threadIdx.x] = dev_a[blockDim.x * blockIdx.x + threadIdx.x];
	__syncthreads();
	if (threadIdx.x < blockDim.x - 2)
	{
		racunaj(dev_niz, dev_b);
	}
	__syncthreads();
}

int main()
{
	int n = 258;
	float* a = (float*)calloc(n, sizeof(float));
	for (int i = 0; i < n; i++)
	{
		a[i] = i;
	}

	float* b = (float*)calloc(n-2, sizeof(float));
	float* dev_a,*dev_b;
	hipMalloc((void**)&dev_a, n*sizeof(float));
	hipMalloc((void**)&dev_b, (n-2)*sizeof(float));
	hipMemcpy(dev_a, a, n*sizeof(float), hipMemcpyHostToDevice);
	dim3 blockDim(n/256+1, 1, 1);
	dim3 threadDim((n/blockDim.x)+1, 1, 1);
	kernel <<<blockDim,threadDim>>> (dev_a,dev_b);
	hipMemcpy(b, dev_b, (n - 2)*sizeof(float), hipMemcpyDeviceToHost);
	for (int i = 0; i < n - 2; i++)
	{
		printf_s("%d,%f \n", i,b[i]);
	}
	hipFree(dev_b);
	hipFree(dev_a);
	free(a);
	free(b);
}
#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include<stdio.h>
#include <stdlib.h>
#include <math.h>
#include ""
#define tile_size 16

__device__ void findMin(int* tajl_C)
{
	for (int i = blockDim.y / 2; i > 0; i >>=1)
	{
		if (threadIdx.y < i)
		{
			if (tajl_C[threadIdx.y * blockDim.x + threadIdx.x] > tajl_C[(threadIdx.y + i) * blockDim.x + threadIdx.x])
			{
				tajl_C[threadIdx.y * blockDim.x + threadIdx.x] = tajl_C[(threadIdx.y + i) * blockDim.x + threadIdx.x];
			}
		}
		__syncthreads();
	}
}

__device__ void izracunaj(int* tajl_A, int* tajl_B,int* dev_C,int tid)
{
	dev_C[tid] = tajl_A[threadIdx.x + threadIdx.y * blockDim.x] -
		2 * tajl_B[threadIdx.y * blockDim.y + threadIdx.x];
}
__global__ void kernel(int* dev_A, int* dev_B, int* dev_C,int n)
{
	__shared__ int tajl_A[tile_size*tile_size], tajl_B[tile_size * tile_size];
	/*int tid = blockIdx.y * blockDim.x * blockDim.y * gridDim.x +
		blockIdx.x*blockDim.x+
		threadIdx.y+threadIdx.x*blockDim.x*gridDim.x;
	int Row = blockIdx.y * tile_size + threadIdx.y;
	int Column = blockIdx.x * tile_size + threadIdx.x;
	int tid = Row * n+ + threadIdx.x;*/

	int tid = blockIdx.y * gridDim.x * blockDim.y * blockDim.x + blockDim.y * blockDim.x * blockIdx.x
		+ threadIdx.y * blockDim.x + threadIdx.x;

	tajl_A[threadIdx.y * blockDim.x + threadIdx.x] = dev_A[tid];
	tajl_B[threadIdx.y * blockDim.x + threadIdx.x] = dev_B[tid];

	__syncthreads();
	if (threadIdx.x + threadIdx.y * blockDim.x < n * n)
		izracunaj(tajl_A, tajl_B, dev_C, tid);
	__syncthreads();

}

__global__ void kernel2(int* dev_C, int* dev_v_extended)
{
	__shared__ int tajl_C[tile_size * tile_size];
	int tid = blockIdx.y * gridDim.x * blockDim.y * blockDim.x + blockDim.x * blockIdx.x
		+ threadIdx.y * blockDim.x * gridDim.x + threadIdx.x;

	tajl_C[threadIdx.y * blockDim.x + threadIdx.x] = dev_C[tid];
	__syncthreads();
	findMin(tajl_C);
	__syncthreads();
	if (threadIdx.y == 0)
	{
		int p = blockIdx.y * blockDim.x * gridDim.x + gridDim.x * blockIdx.x + threadIdx.x;
		dev_v_extended[p] = tajl_C[threadIdx.x];
	}
}
__global__ void kernel3(int* dev_v_extended)
{
	for (int i = blockDim.x / 2; i > 0; i >>= 1)
	{
		if (threadIdx.y < i)
		{
			if (dev_v_extended[threadIdx.y*blockDim.x*gridDim.x+blockIdx.x*blockDim.x+threadIdx.x] 
				> dev_v_extended[threadIdx.y * blockDim.x * gridDim.x + blockIdx.x * blockDim.x + threadIdx.x + i*blockDim.x])
			{
				dev_v_extended[threadIdx.y * blockDim.x * gridDim.x + blockIdx.x * blockDim.x + threadIdx.x] =
					dev_v_extended[threadIdx.y * blockDim.x * gridDim.x + blockIdx.x * blockDim.x + threadIdx.x + i * blockDim.x];
			}
		}
		__syncthreads();
	}
}

int main()
{
	int n = 32;
	int* A=(int*)calloc(n*n,sizeof(int));
	int* B=(int*)calloc(n * n, sizeof(int));
	int* C = (int*)calloc(n * n, sizeof(int));
	int* v = (int*)calloc(n, sizeof(int));
	int* dev_A,*dev_B,*dev_C,* dev_v_extended;

	for (int i = 0; i < n; i++)
	{
		for (int j = 0; j < n; j++)
		{
			A[i * n + j] = i * n + 3*j;
			B[i * n + j] = i * n + j;
		}
	}
	hipMalloc((void**)&dev_A, n*n*sizeof(int));
	hipMalloc((void**)&dev_B, n * n * sizeof(int));
	hipMalloc((void**)&dev_C, n * n * sizeof(int));
	hipMalloc((void**)&dev_v_extended, n* (int)ceil(n / tile_size) *sizeof(int));

	hipMemcpy(dev_A, A, n * n * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_B, B, n * n * sizeof(int), hipMemcpyHostToDevice);
	dim3 blok(tile_size,tile_size,1);
	dim3 grid((int)ceil(n/tile_size),(int)ceil(n/tile_size),1);
	kernel <<<grid,blok>>> (dev_A,dev_B,dev_C,n);
	kernel2 <<<grid,blok>>> (dev_C, dev_v_extended);
	kernel3 <<<dim3((int)ceil(n/tile_size),1,1), dim3(tile_size*tile_size,1,1) >> > (dev_v_extended);
	hipMemcpy(C, dev_C, n * n * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(v, dev_v_extended, n * sizeof(int), hipMemcpyDeviceToHost);

	for (int i = 0; i < n; i++)
	{
		for (int j = 0; j < n; j++)
		{
			printf_s("%d ", C[i * n + j]);
		}
		printf_s("\n");
	}

	for (int i = 0; i < n; i++)
	{
		printf_s("%d ", v[i]);
	}

	hipFree(dev_A);
	hipFree(dev_B);
	hipFree(dev_C);
	hipFree(dev_v_extended);
	free(A);
	free(B);
	free(C);
	free(v);
	return 0;
}
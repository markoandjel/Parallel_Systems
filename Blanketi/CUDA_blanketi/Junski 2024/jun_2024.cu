#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
using namespace std;

#define N 10 
#define Pi 3.14152 

typedef struct krug{
	float x,y,r;
}krug;

__device__ float povrsina(float r)
{
	return r * r * Pi;
}

__global__ void kernel(struct krug* dev_krugovi,struct krug* dev_rezultat)
{
	__shared__ float povrsine[N];
	povrsine[threadIdx.x] = povrsina(dev_krugovi[threadIdx.x].r);
	__syncthreads();
	if (threadIdx.x == 0)
	{
		float pom_max = povrsine[0];
		int index_max = 0;
		float pom = povrsine[0];
		int index = 0;
		for (int i = 1; i < N; i++)
		{
			if (povrsine[i] > pom_max)
			{
				pom = pom_max;
				index = index_max;
				pom_max = povrsine[i];
				index_max = i;
			}
			else if(povrsine[i]>pom)
			{
				pom = povrsine[i];
				index = i;
			}
		}
		dev_rezultat[0].r = dev_krugovi[index].r;
		dev_rezultat[0].x = dev_krugovi[index].x;
		dev_rezultat[0].y = dev_krugovi[index].y;

		dev_rezultat[1].r = dev_krugovi[index_max].r;
		dev_rezultat[1].x = dev_krugovi[index_max].x;
		dev_rezultat[1].y = dev_krugovi[index_max].y;
	}
}

int main()
{
	krug rezultat[2];
	krug krugovi[N];
	for (int i = 0; i < N; i++)
	{
		krugovi[i].x = i * N;
		krugovi[i].y = i + N;
		krugovi[i].r = i + 1;
	}
	krug* dev_krugovi;
	krug* dev_rezultat;
	//printf_s("%d,%d\n", (int)sizeof(struct krug),sizeof(double));
	hipMalloc((void**)&dev_krugovi, N * sizeof(krug));
	hipMemcpy(dev_krugovi, &krugovi[0], N * sizeof(krug), hipMemcpyHostToDevice);
	hipMalloc((void**)&dev_rezultat, 2*sizeof(krug));
	kernel <<<1, N>>> (dev_krugovi, dev_rezultat);

	hipMemcpy(&rezultat, dev_rezultat, 2*sizeof(krug), hipMemcpyDeviceToHost);
	hipFree(dev_krugovi);
	hipFree(dev_rezultat);
	printf_s("%f,%f:%f \n", rezultat[0].x, rezultat[0].y, rezultat[0].r);
	printf_s("%f,%f:%f \n",rezultat[1].x, rezultat[1].y, rezultat[1].r);
}
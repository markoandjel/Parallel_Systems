#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include ""
#include "math.h"
using namespace std;

#define blockSize 32


__global__ void kernel(int* dev_A,int*dev_B,int*dev_C,int n)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < n * n)
	{
		int row = i / n;
		int col = i % n;
		int ind = col * n + row;
		dev_C[i] = dev_A[i] < dev_B[ind] ? dev_A[i] : dev_B[ind];
		i += blockIdx.x * blockDim.x;
	}
}

__global__ void kernel2(int* dev_C, float* dev_rez,int n)
{
	__shared__ int suma[blockSize];
	//int slic = n % blockDim.x == 0 ? (int) n / blockDim.x : (int)n/blockDim.x + 1;
	int brojBloka = blockIdx.x;
	while (brojBloka<n)
	{
		/*int start = threadIdx.x * slic;
		int end = start + slic;
		if (end > n) end = n;
		for (int i = start; i < end; i++)
		{
			if (dev_C[brojBloka * n + i] < localMin) localMin = dev_C[brojBloka * n + i];
		}
		*/
		suma[threadIdx.x] = dev_C[threadIdx.x * blockDim.x + blockIdx.x * blockDim.x];
		__syncthreads();
		for (int s = n / 2; s > 0; s /= 2)
		{
			if (threadIdx.x < s)
			{
				suma[threadIdx.x] += suma[threadIdx.x + s];
			}
			__syncthreads();
		}
		if (threadIdx.x == 0)
		{
			dev_rez[brojBloka] = (float)suma[0]/n;
		}
		brojBloka += gridDim.x;
	}
}

int main()
{
	int n = 16;
	int* A = (int*)calloc(n * n, n * n * sizeof(int));
	int* B = (int*)calloc(n * n, n * n * sizeof(int));
	int* C = (int*)calloc(n * n, n * n * sizeof(int));
	float* rez = (float*)calloc(n, n * sizeof(float));
	for (int i = 0; i < n * n; i++)
	{
		A[i] = i + 1;
		B[i] = i+2;
	}
	int* dev_A, * dev_B, * dev_C;
	float*dev_rez;
	hipMalloc((void**)&dev_A, n * n * sizeof(int));
	hipMalloc((void**)&dev_B, n * n * sizeof(int));
	hipMalloc((void**)&dev_C, n * n * sizeof(int));
	hipMalloc((void**)&dev_rez,n * sizeof(float));

	hipMemcpy(dev_A, A, n * n * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_B, B, n * n * sizeof(int), hipMemcpyHostToDevice);

	kernel <<<blockSize, blockSize >> > (dev_A,dev_B,dev_C,n);
	hipMemcpy(C, dev_C, n * n * sizeof(int), hipMemcpyDeviceToHost);

	kernel2 << <blockSize, n >> > (dev_C, dev_rez,n);
	hipMemcpy(rez, dev_rez, n * sizeof(float), hipMemcpyDeviceToHost);

	for (int i = 0; i < n; i++)
	{
		for (int j = 0; j < n; j++)
		{
			printf_s("%d ", C[i * n + j]);
		}
		printf_s("\n");
	}

	for (int i = 0; i < n; i++)
	{
		printf_s("%.4f ", rez[i]);
	}

	free(A); free(B); free(C);
}
#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include<stdio.h>
#include <math.h>
#include ""
#define N 128
#define FLT_MAX 3.402823466e+38F
typedef struct pov
{
	float rastojanje; int index;
}pov;
typedef struct point
{
	float x, y;
}point;
 __device__ float rastojanje(point* t1, point* t2)
{
	return sqrtf((t1->x - t2->x) * (t1->x - t2->x) + (t1->y - t2->y) * (t1->y - t2->y));
}

 __device__ void findMin(pov* dev_rastind)
 {
	for (unsigned int offset = blockDim.x / 2; offset > 0; offset >>= 1)
	{
		if (threadIdx.x < offset) {
			if (dev_rastind[threadIdx.x].rastojanje > dev_rastind[threadIdx.x + offset].rastojanje)
			{
				dev_rastind[threadIdx.x].rastojanje = dev_rastind[threadIdx.x + offset].rastojanje;
				dev_rastind[threadIdx.x].index = dev_rastind[threadIdx.x + offset].index;
			}
		}
		__syncthreads();
	}
 }

__global__ void kernel(point* dev_niz,point*tacka,point* dev_blizu)
{
	__shared__ float rastojanja[N];
	__shared__ pov dev_rastind[N];
	__shared__ int indeksi[10];
	if (threadIdx.x < 10)
	{
		indeksi[threadIdx.x] = -1;
	}
	rastojanja[threadIdx.x] = rastojanje(tacka, &dev_niz[threadIdx.x]);
	/*dev_rastind[threadIdx.x].rastojanje = rastojanja[threadIdx.x];
	dev_rastind[threadIdx.x].index = threadIdx.x;*/
	__syncthreads();

	for (int i = 0; i < 10; i++)
	{
		dev_rastind[threadIdx.x].rastojanje = rastojanja[threadIdx.x];
		dev_rastind[threadIdx.x].index = threadIdx.x;
		__syncthreads();
		if (threadIdx.x < 10)
		{
			if (indeksi[threadIdx.x] > -1)
			{
				dev_rastind[indeksi[threadIdx.x]].rastojanje = FLT_MAX;
			}
		}
		__syncthreads();

		findMin(dev_rastind);
		__syncthreads();
		if(threadIdx.x==0)
		indeksi[i] = dev_rastind[0].index;
	}
	__syncthreads();
	if (threadIdx.x<10)
	{
		dev_blizu[threadIdx.x].x = dev_niz[indeksi[threadIdx.x]].x;
		dev_blizu[threadIdx.x].y = dev_niz[indeksi[threadIdx.x]].y;
	}

}
int main()
{
	point niz[N],blizu[10];
	point tacka;
	for (int i = 0; i < N; i++)
	{
		niz[i].x = i + 1.54;
		niz[i].y = i - 1.32;
	}
	//tacka.x = 5.543;
	//tacka.y = 3.658;
	printf_s("Unesite x:\n");
	scanf_s("%f", &tacka.x);
	printf_s("Unesite y:\n");
	scanf_s("%f", &tacka.y);

	point* dev_niz,*dev_tacka,*dev_blizu;
	hipMalloc((void**)&dev_niz, N * sizeof(point));
	hipMalloc((void**)&dev_tacka, sizeof(point));
	hipMalloc((void**)&dev_blizu, 10*sizeof(point));
	hipMemcpy(dev_niz, niz, N * sizeof(point), hipMemcpyHostToDevice);
	hipMemcpy(dev_tacka, &tacka, sizeof(point), hipMemcpyHostToDevice);

	kernel<<<1,N>>>(dev_niz, dev_tacka, dev_blizu);

	hipMemcpy(blizu, dev_blizu, 10*sizeof(point), hipMemcpyDeviceToHost);
	hipFree(dev_niz);
	hipFree(dev_tacka);
	hipFree(blizu);
	for (int i = 0; i < 10; i++)
	{
		printf_s("%f,%f\n", blizu[i].x, blizu[i].y);
	}
	return 0;
}
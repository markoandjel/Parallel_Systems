#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include ""
#include "math.h"

#define block_size 256

__global__ void kernel(int* A_dev, int* B_dev, int* C_dev,int n)
{
	int ind = blockIdx.x * blockDim.x + threadIdx.x;
	if (ind < n * n)
	{
		int r = ind / n;
		int c = ind % n;
		int ind2 = c * n + r;
		if (A_dev[ind] > B_dev[ind2])
			C_dev[ind] = A_dev[ind];
		else C_dev[ind] = B_dev[ind2];
	}
}


__global__ void kernel2(int* C_dev,int n,int* v_dev)
{
	__shared__ int local_v[block_size];
	if (threadIdx.x < n)
	{
		local_v[threadIdx.x] = C_dev[n * blockIdx.x + threadIdx.x];
	}
	__syncthreads();

	if (threadIdx.x >= n) local_v[threadIdx.x] = INT_MAX;

	__syncthreads();
	for (int s = blockDim.x / 2; s > 0; s /= 2)
	{
		if (threadIdx.x < s)
		{
			if (local_v[threadIdx.x] > local_v[threadIdx.x + s])
			{
				local_v[threadIdx.x] = local_v[threadIdx.x + s];
			}
		}
		__syncthreads();
	}
	if (threadIdx.x == 0)
	{
		v_dev[blockIdx.x] = local_v[0];
	}

}


int main()
{
	int n = 50;
	int* A = (int*)calloc(n * n, n * n * sizeof(int));
	int* B = (int*)calloc(n * n, n * n * sizeof(int));
	int* C = (int*)calloc(n * n, n * n * sizeof(int));
	int* v = (int*)calloc(n, n * sizeof(int));

	int* A_dev = (int*)calloc(n * n, n * n * sizeof(int));
	int* B_dev = (int*)calloc(n * n, n * n * sizeof(int));
	int* C_dev = (int*)calloc(n * n, n * n * sizeof(int));
	int* v_dev = (int*)calloc(n, n * sizeof(int));


	for (int i = 0; i < n * n; i++)
	{
		A[i] = i;
		B[i] = i + 1;
	}

	hipMalloc((void**)&A_dev, n * n * sizeof(int));
	hipMalloc((void**)&B_dev, n * n * sizeof(int));
	hipMalloc((void**)&C_dev, n * n * sizeof(int));
	hipMalloc((void**)&v_dev, n * sizeof(int));

	hipMemcpy(A_dev, A, n * n * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(B_dev, B, n * n * sizeof(int), hipMemcpyHostToDevice);

	kernel <<< block_size, block_size >>> (A_dev, B_dev, C_dev,n);
	kernel2 << <block_size, block_size >> > (C_dev, n, v_dev);



	hipMemcpy(C, C_dev, n * n * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(v, v_dev, n * sizeof(int), hipMemcpyDeviceToHost);

	for (int i = 0; i < n; i++)
	{
		for (int j = 0; j < n; j++)
		{
			printf_s("%d ", C[i * n + j]);
		}
		printf_s("\n");
	}

	for (int i = 0; i < n; i++)
	{
		printf_s("%d ", v[i]);
	}



	free(A); free(B); free(C); free(v);
	hipFree(A_dev); hipFree(B_dev); hipFree(C_dev); hipFree(v_dev);
	return 0;
}